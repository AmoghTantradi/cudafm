#include "hip/hip_runtime.h"
#include "./util/fmatrix.h"
#include "data.h"
#include "fm.h"
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

// =================
// Helper Functions
// =================

// Command Line Option Processing

// ==============
// Main Function
// ==============

int main(int argc, char** argv) {
    // const std::string param_train_file	= "../data/ml-tag.test.libfm"; // "libfm_test.txt";
    // // const std::string param_train_file	= "../scripts/libfm_test_data_large.txt"; //
    // "libfm_test.txt"; Data train; train.load(param_train_file); for (int i = 0; i < 10; i++) {
    // 	std :: cout << train.target[i] << " ";
    // 	for (int j = 0; j < train.data[i]->size; j++) {
    // 		std::cout << train.data[i]->data[j].id << ":" << train.data[i]->data[j].value << "
    // ";
    // 	}
    // 	std::cout << std :: endl;
    // }
    // fm_model fm(train.num_feature, 8);
    // fm.params.learn_rate = 0.05;
    // fm.params.task = 1;
    // fm.params.min_target = train.min_target;
    // fm.params.max_target = train.max_target;
    auto start_time = std::chrono::steady_clock::now();
    // fm.learn(&train, &train, 2);

    // [[1 0 2]
    // [0 3 0]
    // [4 0 5]]

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    double values[5] = {1, 2, 3, 4, 5};
    int colIdx[5] = {0, 2, 1, 0, 2};
    int rowPtr[4] = {0, 2, 3, 5};

    double* d_A_values;
    hipMalloc((void**)&d_A_values, 5 * sizeof(double));
    hipMemcpy(d_A_values, values, 5 * sizeof(double), hipMemcpyHostToDevice);

    hipsparseSpMatDescr_t descrA;
    hipsparseCreateCsr(&descrA, 3, 3, 5, rowPtr, colIdx, d_A_values, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    double* a_C = (double*)malloc(5 * sizeof(double));

    hipMemcpy(a_C, d_A_values, 5 * sizeof(double), hipMemcpyDeviceToHost);

    double values2[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};

    hipsparseDnMatDescr_t descrB;

    double* d_B_values;
    hipMalloc((void**)&d_B_values, 9 * sizeof(double));
    hipMemcpy(d_B_values, values2, 9 * sizeof(double), hipMemcpyHostToDevice);

    hipsparseCreateDnMat(&descrB, 3, 3, 3, d_B_values, HIP_R_64F, HIPSPARSE_ORDER_ROW);

    double* b_C = (double*)malloc(9 * sizeof(double));

    hipMemcpy(b_C, d_B_values, 9 * sizeof(double), hipMemcpyDeviceToHost);

    double* d_values;
    int* d_colIdx;
    int* d_rowPtr;

    hipMalloc((void**)&d_values, 5 * sizeof(double));
    hipMalloc((void**)&d_colIdx, 5 * sizeof(int));
    hipMalloc((void**)&d_rowPtr, 4 * sizeof(int));

    hipMemcpy(d_values, values, 5 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, colIdx, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtr, rowPtr, 4 * sizeof(int), hipMemcpyHostToDevice);


    // double* d_C;
    // hipMalloc((void**)&d_C, 9 * sizeof(double));

    // size_t buffer_size;

    // int nnzC = 0;
    // int* nnzTotalDevHostPtr = &nnzC;

    // const float alpha = 1.0;
    // const float beta = 0.0;
    /*
    hipsparseSpMMAlg_t alg = HIPSPARSE_MM_ALG_DEFAULT;

    hipsparseDnMatDescr_t descrC;

    double valuesC[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};

    double* d_C_values;
    hipMalloc((void**)&d_C_values, 9 * sizeof(double));
    hipMemcpy(d_C_values, valuesC, 9 * sizeof(double), hipMemcpyHostToDevice);

    hipsparseCreateDnMat(&descrC, 3, 3, 3, d_C_values, HIP_R_64F, HIPSPARSE_ORDER_ROW);
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, (void*)&alpha, descrA, descrB,
                            (void*)&beta, descrC, HIP_R_64F, alg, &buffer_size);

    hipDeviceSynchronize();

    void* buffer = 0;
    hipMalloc(&buffer, buffer_size);

    hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 (void*)&alpha, descrA, descrB, (void*)&beta, descrC, HIP_R_64F, alg, buffer);
   */
   
   
   
    // double valuesC[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};
  
   
    // hipsparseDnMatDescr_t descrC;


    



    // double* h_C = (double*)malloc(9 * sizeof(double));
    // hipMemcpy(h_C, &descrC, 9 * sizeof(double), hipMemcpyDeviceToHost);



    // std::cout << "Result: " << std::endl;
    // for (int i = 0; i < 9; i++) {
    //     std::cout << h_C[i] << " ";
    // }

// works somewhat
	hipsparseDnMatDescr_t descrC;

	double valuesC[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};

	double* d_C_values;
	hipMalloc((void**)&d_C_values, 9 * sizeof(double));
	hipMemcpy(d_C_values, valuesC, 9 * sizeof(double), hipMemcpyHostToDevice);

	hipsparseCreateDnMat(&descrC, 3, 3, 3, d_C_values, HIP_R_64F, HIPSPARSE_ORDER_ROW);

	std::cout << "Result: " << std::endl;


       double* valuesdv;
       int64_t rows;
       int64_t cols;
       int64_t ld;
       hipDataType cuda_data_type;
       hipsparseOrder_t order;


        hipsparseDnMatGet(
                descrC,
                &rows, 
                &cols, 
                &ld, 
                (void **) &valuesdv,
                &cuda_data_type,
                &order			                
        );





	double* h_C = new double[9];
        


	hipMemcpy(h_C, valuesdv, 9 * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < 9; i++) {
		std::cout << h_C[i] << " ";
	}
	delete[] h_C;
	hipFree(d_C_values);




    // free(h_C);
    // hipFree(d_values);
    // hipFree(d_colIdx);
    // hipFree(d_rowPtr);
    // // hipFree(d_C);
    // hipsparseDestroySpMat(descrA);
    // hipsparseDestroyDnMat(descrB);
    // hipsparseDestroyDnMat(descrC);
    // hipsparseDestroy(handle);

    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();
    // Finalize
    std::cout << "Simulation Time = " << seconds << " seconds \n";
    /*
srand ( time(NULL) );
    try {

            const std::string param_task		= "c";
            const std::string param_train_file	= "libfm_test.txt";
            const std::string param_test_file	= "libfm_test.txt";

            double param_init_stdev	= 0.1;
            int param_num_iter	= 100;
            double param_learn_rate	= 0.01;
            const std::string param_method		= "sgd";

            const std::string param_do_sampling	= "do_sampling";
            const std::string param_do_multilevel	= "do_multilevel";
            const std::string param_num_eval_cases  = "num_eval_cases";
            // (1) Load the data
            std::cout << "Loading train...\t" << std::endl;
            Data train(
                    0,
                    ! (!param_method.compare("mcmc")), // no original data for mcmc
                    ! (!param_method.compare("sgd") || !param_method.compare("sgda")) // no
transpose data for sgd, sgda
            );
            train.load(param_train_file);

            std::cout << "Loading test... \t" << std::endl;
            Data test(
                    0,
                    ! (!param_method.compare("mcmc")), // no original data for mcmc
                    ! (!param_method.compare("sgd") || !param_method.compare("sgda")) // no
transpose data for sgd, sgda
            );
            test.load(param_test_file);

            Data* validation = NULL;

            // (2) Setup the factorization machine
            fm_model fm;
            {
        uint num_all_attribute = std::max(train.num_feature, test.num_feature);
                    fm.num_attribute = num_all_attribute;
                    fm.init_stdev = param_init_stdev;
                    // set the number of dimensions in the factorization
                    {
                            std::vector<int> dim(3);
            dim[0] = 1;
            dim[1] = 1;
            dim[2] = 8;
                            assert(dim.size() == 3);
                            fm.k0 = dim[0] != 0;
                            fm.k1 = dim[1] != 0;
                            fm.num_factor = dim[2];
                    }
                    fm.init();

            }

            // (3) Setup the learning method:
            fm_learn* fml;
            if (! param_method.compare("sgd")) {
                    fml = new fm_learn();
                    fml->num_iter = param_num_iter;
            } else {
                    throw "unknown method";
            }
            fml->fm = &fm;
            fml->max_target = train.max_target;
            fml->min_target = train.min_target;
            if (! param_task.compare("r") ) {
                    fml->task = 0;
            } else if (! param_task.compare("c") ) {
                    fml->task = 1;
                    for (uint i = 0; i < train.target.dim; i++) { if (train.target(i) <= 0.0) {
train.target(i) = -1.0; } else {train.target(i) = 1.0; } } for (uint i = 0; i < test.target.dim;
i++) { if (test.target(i) <= 0.0) { test.target(i) = -1.0; } else {test.target(i) = 1.0; } } } else
{ throw "unknown task";
            }

    fml->init();
    // set the regularization; for standard SGD, groups are not supported
    {
        std::vector<double> reg(3);
        reg[2] == 0.01;
        assert((reg.size() == 0) || (reg.size() == 1) || (reg.size() == 3));
        if (reg.size() == 0) {
            fm.reg0 = 0.0;
            fm.regw = 0.0;
            fm.regv = 0.0;
        } else if (reg.size() == 1) {
            fm.reg0 = reg[0];
            fm.regw = reg[0];
            fm.regv = reg[0];
        } else {
            fm.reg0 = reg[0];
            fm.regw = reg[1];
            fm.regv = reg[2];
        }
    }
            {
                    std::vector<double> lr(1, param_learn_rate);
                    assert((lr.size() == 1) || (lr.size() == 3));
                    if (lr.size() == 1) {
                            fml->learn_rate = lr[0];
                            fml->learn_rates.init(lr[0]);
                    } else {
                            fml->learn_rate = 0;
                            fml->learn_rates(0) = lr[0];
                            fml->learn_rates(1) = lr[1];
                            fml->learn_rates(2) = lr[2];
                    }
            }

            auto start_time = std::chrono::steady_clock::now();

            // () learn
            fml->learn(&train, &test);

            // () Prediction at the end  (not for mcmc and als)
            std::cout << "Final\t" << "Train=" << fml->evaluate(&train) << "\tTest=" <<
fml->evaluate(&test) << std::endl;

            auto end_time = std::chrono::steady_clock::now();
            std::chrono::duration<double> diff = end_time - start_time;
            double seconds = diff.count();

            // Finalize
            std::cout << "Simulation Time = " << seconds << " seconds \n";


    } catch (std::string &e) {
            std::cerr << std::endl << "ERROR: " << e << std::endl;
    } catch (char const* &e) {
            std::cerr << std::endl << "ERROR: " << e << std::endl;
    }
    */
}
