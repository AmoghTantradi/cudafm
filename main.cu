#include "./util/fmatrix.h"
#include "data.h"
#include "fm.h"
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

// =================
// Helper Functions
// =================

// Command Line Option Processing

// ==============
// Main Function
// ==============


void printCudaSparse(hipsparseSpMatDescr_t sparse_descr) {
  double* values_dev;
  int32_t* row_indices_dev;
  int32_t* col_indices_dev;
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  hipsparseIndexType_t rowidx_type;
  hipsparseIndexType_t colidx_type;
  hipsparseIndexBase_t idx_base;
  hipDataType cuda_data_type;

  hipsparseCsrGet(
    sparse_descr,
    &rows,
    &cols,
    &nnz,
    (void**)&row_indices_dev,
    (void**)&col_indices_dev,
    (void**)&values_dev,
    &rowidx_type,
    &colidx_type,
    &idx_base,
    &cuda_data_type
  );
  double * values_host = new double[nnz];
  int32_t* row_indices_host = new int32_t[nnz];
  int32_t* col_indices_host = new int32_t[nnz];
  hipMemcpy(values_host, values_dev, nnz*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(row_indices_host, row_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);
  hipMemcpy(col_indices_host, col_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);
  std::cout << "nnz: " << nnz << " " << std::endl;
  
  for (int64_t i = 0 ; i < nnz; i++) {
        std::cout << values_host[i] << " " ;
        std::cout << ": " << col_indices_host[i] << " ";
  }
  std::cout << std::endl;
  delete [] values_host;
}


/*
void printCudaDense(hipsparseDnMatDescr_t descrC) {
    double* valuesdv;
    int64_t rows;
    int64_t cols;
    int64_t ld;
    hipDataType cuda_data_type;
    hipsparseOrder_t order;
    hipsparseDnMatGet(descrC, &rows, &cols, &ld, (void**)&valuesdv, &cuda_data_type, &order);
    double* h_C = new double[9];
    hipMemcpy(h_C, valuesdv, 9 * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Result: " << std::endl;
    for (int i = 0; i < 9; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;
    delete[] h_C;
}
*/

int main(int argc, char** argv) {
    auto start_time = std::chrono::steady_clock::now();
    
        // Host problem definition
    int   A_num_rows      = 4;
    int   A_num_cols      = 4;
    int   A_nnz           = 9;
    int   B_num_rows      = A_num_cols;
    int   B_num_cols      = 3;
    int   ldb             = B_num_rows;
    int   ldc             = A_num_rows;
    int   B_size          = ldb * B_num_cols;
    int   C_size          = ldc * B_num_cols;
    int   hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
    int   hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    float hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                              6.0f, 7.0f, 8.0f, 9.0f };
    float hB[]            = { 1.0f,  2.0f,  3.0f,  4.0f,
                              5.0f,  6.0f,  7.0f,  8.0f,
                              9.0f, 10.0f, 11.0f, 12.0f };
    float hC[]            = { 0.0f, 0.0f, 0.0f, 0.0f,
                              0.0f, 0.0f, 0.0f, 0.0f,
                              0.0f, 0.0f, 0.0f, 0.0f };
    float hC_result[]     = { 19.0f,  8.0f,  51.0f,  52.0f,
                              43.0f, 24.0f, 123.0f, 120.0f,
                              67.0f, 40.0f, 195.0f, 188.0f };
    float alpha           = 1.0f;
    float beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dB, *dC;
    ( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) );
    ( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))    );
    ( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))  );
    ( hipMalloc((void**) &dB,         B_size * sizeof(float)) );
    ( hipMalloc((void**) &dC,         C_size * sizeof(float)) );

    ( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) );
    ( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) );
    ( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                           hipMemcpyHostToDevice) );
    ( hipMemcpy(dB, hB, B_size * sizeof(float),
                           hipMemcpyHostToDevice) );
    ( hipMemcpy(dC, hC, C_size * sizeof(float),
                           hipMemcpyHostToDevice) );
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    ( hipsparseCreate(&handle) );
    // Create sparse matrix A in CSR format
    ( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
    // Create dense matrix B
    ( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) );
    // Create dense matrix C
    ( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) );
    // allocate an external buffer if needed
    ( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) );
    ( hipMalloc(&dBuffer, bufferSize) );

    // execute SpMM
    ( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) );

  
    //--------------------------------------------------------------------------
    // device result check
    ( hipMemcpy(hC, dC, C_size * sizeof(float),
                           hipMemcpyDeviceToHost) );
    int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        for (int j = 0; j < B_num_cols; j++) {
            if (hC[i + j * ldc] != hC_result[i + j * ldc]) {
                correct = 0; // direct floating point comparison is not reliable
                break;
            }
        }
    }
    if (correct)
        printf("spmm_csr_example test PASSED\n");
    else
        printf("spmm_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------





    //now modify dB

    float* temp = (float* ) malloc(sizeof(float) * B_size);

    hipMemcpy(temp, dB, sizeof(float) * B_size, hipMemcpyDeviceToHost);

    temp[0] = 10.0f; //changing from 1.0f to 10.0f
    temp[1] = -20.0f; //changing from 2.0f to -20.0f

    hipMemcpy(dB, temp, sizeof(float) * B_size, hipMemcpyHostToDevice); // changing and copying the values back 

    //now perform the spmm again

    int correct2 = 1;

    ( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) );
    ( hipMalloc(&dBuffer, bufferSize) );

    // execute SpMM
    ( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) );



    ( hipMemcpy(hC, dC, C_size * sizeof(float),
                           hipMemcpyDeviceToHost) );

    for (int i = 0; i < A_num_rows; i++) {
        for (int j = 0; j < B_num_cols; j++) {
            if (hC[i + j * ldc] != hC_result[i + j * ldc]) {
                correct2 = 0; // direct floating point comparison is not reliable
                break;
            }
        }
    }
    if (correct2)
        printf("spmm_csr_example test 2 PASSED\n");
    else
        printf("spmm_csr_example test 2 FAILED: wrong result\n"); //what we want to happen








      // destroy matrix/vector descriptors
    ( hipsparseDestroySpMat(matA) );
    ( hipsparseDestroyDnMat(matB) );
    ( hipsparseDestroyDnMat(matC) );
    ( hipsparseDestroy(handle) );


    // device memory deallocation
    ( hipFree(dBuffer) );
    ( hipFree(dA_csrOffsets) );
    ( hipFree(dA_columns) );
    ( hipFree(dA_values) );
    ( hipFree(dB) );
    ( hipFree(dC) );

    auto end_time = std::chrono::steady_clock::now();

	std::chrono::duration<double> diff = end_time - start_time;
	double seconds = diff.count();

		// Finalize
	std::cout << "Simulation Time = " << seconds << " seconds \n";
}
