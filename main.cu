#include "hip/hip_runtime.h"
#include "./util/fmatrix.h"
#include "data.h"
#include "fm.h"
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

// =================
// Helper Functions
// =================

// I/O routines
void save(std::ofstream& fsave, particle_t* parts, int num_parts, double size) {
    static bool first = true;

    if (first) {
        fsave << num_parts << " " << size << std::endl;
        first = false;
    }

    for (int i = 0; i < num_parts; ++i) {
        fsave << parts[i].x << " " << parts[i].y << std::endl;
    }

    fsave << std::endl;
}


/*
// Particle Initialization
void init_particles(particle_t* parts, int num_parts, double size, int part_seed) {
    std::random_device rd;
    std::mt19937 gen(part_seed ? part_seed : rd());

    int sx = (int)ceil(sqrt((double)num_parts));
    int sy = (num_parts + sx - 1) / sx;

    std::vector<int> shuffle(num_parts);
    for (int i = 0; i < shuffle.size(); ++i) {
        shuffle[i] = i;
    }

    for (int i = 0; i < num_parts; ++i) {
        // Make sure particles are not spatially sorted
        std::uniform_int_distribution<int> rand_int(0, num_parts - i - 1);
        int j = rand_int(gen);
        int k = shuffle[j];
        shuffle[j] = shuffle[num_parts - i - 1];

        // Distribute particles evenly to ensure proper spacing
        parts[i].x = size * (1. + (k % sx)) / (1 + sx);
        parts[i].y = size * (1. + (k / sx)) / (1 + sy);

        // Assign random velocities within a bound
        std::uniform_real_distribution<float> rand_real(-1.0, 1.0);
        parts[i].vx = rand_real(gen);
        parts[i].vy = rand_real(gen);
    }
}
*/


// Command Line Option Processing

// ==============
// Main Function
// ==============


void printCudaSparse(hipsparseSpMatDescr_t sparse_descr) {
  double* values_dev;
  int32_t* row_indices_dev;
  int32_t* col_indices_dev;
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  hipsparseIndexType_t rowidx_type;
  hipsparseIndexType_t colidx_type;
  hipsparseIndexBase_t idx_base;
  hipDataType cuda_data_type;

  hipsparseCsrGet(
    sparse_descr,
    &rows,
    &cols,
    &nnz,
    (void**)&row_indices_dev,
    (void**)&col_indices_dev,
    (void**)&values_dev,
    &rowidx_type,
    &colidx_type,
    &idx_base,
    &cuda_data_type
  );
  double * values_host = new double[nnz];
  int32_t* row_indices_host = new int32_t[nnz];
  int32_t* col_indices_host = new int32_t[nnz];
  hipMemcpy(values_host, values_dev, nnz*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(row_indices_host, row_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);
  hipMemcpy(col_indices_host, col_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);
  std::cout << "nnz: " << nnz << " " << std::endl;
  
  for (int64_t i = 0 ; i < nnz; i++) {
        std::cout << values_host[i] << " " ;
        std::cout << ": " << col_indices_host[i] << " ";
  }
  std::cout << std::endl;
  delete [] values_host;
}

void printCudaDense(hipsparseDnMatDescr_t descrC) {
    double* valuesdv;
    int64_t rows;
    int64_t cols;
    int64_t ld;
    hipDataType cuda_data_type;
    hipsparseOrder_t order;
    hipsparseDnMatGet(descrC, &rows, &cols, &ld, (void**)&valuesdv, &cuda_data_type, &order);
    double* h_C = new double[9];
    hipMemcpy(h_C, valuesdv, 9 * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Result: " << std::endl;
    for (int i = 0; i < 9; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;
    delete[] h_C;
}

int main(int argc, char** argv) {

    // construct a fm object
    const std::string param_train_file	= "../data/ml-tag.train.libfm"; // "libfm_test.txt";
    // const std::string param_train_file	= "../scripts/libfm_test_data_large.txt"; //
    //"libfm_test.txt";
    Data train;
    train.load(param_train_file);
    /*
    for (int i = 0; i < 10; i++) {
    	std :: cout << train.target[i] << " ";
    	for (int j = 0; j < train.data[i]->size; j++) {
    		std::cout << train.data[i]->data[j].id << ":" << train.data[i]->data[j].value << "\n";
    	}
    }
    */

    std::cout << "Num feature " << train.num_feature << std::endl;
    fm_model fm(train.num_feature, 8);
    fm.params.learn_rate = 0.05;
    fm.params.task = 1;
    fm.params.min_target = train.min_target;
    fm.params.max_target = train.max_target;
    std::vector<trainBatch> batches;
    //std::default_random_engine rng(std::chrono::system_clock::now().time_since_epoch().count());
    //std::shuffle(train.data.begin(), train.data.end(), rng);
    fm.batchSamples(&train, batches);
    std::cout << batches.size() << " batches for " << train.data.size() << " samples \n";
    auto start_time = std::chrono::steady_clock::now();
    fm.learn(batches, 1); 

    /*
    for (int i = 0; i < 1; i++) {
        
        std::cout << "here" << std::endl;
        printCudaSparse(batches[i].x);

        //printCudaSparse(batches[i].second);
        
        // make result matrix
        hipsparseDnMatDescr_t result;
        // have to allocate device memory too

        double* values_dev;
        int32_t* row_indices_dev;
        int32_t* col_indices_dev;
        int64_t rows;
        int64_t cols;
        int64_t nnz;
        hipsparseIndexType_t rowidx_type;
        hipsparseIndexType_t colidx_type;
        hipsparseIndexBase_t idx_base;
        hipDataType cuda_data_type;

        //std::cout << "here1: " << nnz << std::endl;
        hipsparseCsrGet(
            batches[i].x,
            &rows,
            &cols,
            &nnz,
            (void**)&row_indices_dev,
            (void**)&col_indices_dev,
            (void**)&values_dev,
            &rowidx_type,
            &colidx_type,
            &idx_base,
            &cuda_data_type
        );

        double* host = (double*) malloc(sizeof(double) * rows * 8);
        double* devptr;
        //std::cout << rows << " " << cols << "\n";
        
        hipMalloc((void**)&devptr, sizeof(double) * rows * 8); 

        //hipMemcpy(devptr, host, sizeof(double) * rows * 8, hipMemcpyHostToDevice);
        //std::cout << "here2: " << rows << std::endl;
        //std::cout << "here3: " << cols << std::endl;
        hipsparseCreateDnMat(&result, rows, 8, cols, devptr, HIP_R_64F, HIPSPARSE_ORDER_ROW); 
        //std::cout << "created dn\n";
        fm.matMul(batches[i].x, fm.V, result);


 
    //   break;
    }
*/




    //device memory deallocation
    //hipFree(dBuffer);
    // hipFree(dA_csrOffsets);
    // hipFree(dA_columns);
    // hipFree(dA_values);
    // hipFree(dB);
    // hipFree(dC);
    

    auto end_time = std::chrono::steady_clock::now();
	std::chrono::duration<double> diff = end_time - start_time;
	double seconds = diff.count();

		// Finalize
	std::cout << "Total Simulation Time for SGD = " << seconds << " seconds \n";
    std::cout << "Simulation Time for predict" << fm.predictTime << std::endl;
    std::cout << "Time for SGD without predict" << seconds - fm.predictTime << std::endl;
}
