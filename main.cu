#include "hip/hip_runtime.h"
#include "./util/fmatrix.h"
#include "data.h"
#include "fm.h"
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

// =================
// Helper Functions
// =================

// Command Line Option Processing

// ==============
// Main Function
// ==============

/*
void printCudaSparse(hipsparseSpMatDescr_t sparse_descr) {
  double* values_dev;
  int32_t* row_indices_dev;
  int32_t* col_indices_dev;
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  hipsparseIndexType_t rowidx_type;
  hipsparseIndexType_t colidx_type;
  hipsparseIndexBase_t idx_base;
  hipDataType cuda_data_type;

  hipsparseCsrGet(
    sparse_descr,
    &rows,
    &cols,
    &nnz,
    (void**)&row_indices_dev,
    (void**)&col_indices_dev,
    (void**)&values_dev,
    &rowidx_type,
    &colidx_type,
    &idx_base,
    &cuda_data_type
  );
  double * values_host = new double[nnz];
  int32_t* row_indices_host = new int32_t[nnz];
  int32_t* col_indices_host = new int32_t[nnz];
  hipMemcpy(values_host, values_dev, nnz*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(row_indices_host, row_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);
  hipMemcpy(col_indices_host, col_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);
  std:: cout << "Host values " << std::endl;
  for (int64_t i = 0 ; i < nnz; i++) {
        std::cout << values_host[i] << std::endl;
  }
  std::cout << std::endl;
  delete [] values_host;
}

void printCudaDense(hipsparseDnMatDescr_t descrC) {
    double* valuesdv;
    int64_t rows;
    int64_t cols;
    int64_t ld;
    hipDataType cuda_data_type;
    hipsparseOrder_t order;
    hipsparseDnMatGet(descrC, &rows, &cols, &ld, (void**)&valuesdv, &cuda_data_type, &order);
    double* h_C = new double[9];
    hipMemcpy(h_C, valuesdv, 9 * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Result: " << std::endl;
    for (int i = 0; i < 9; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;
    delete[] h_C;
}
*/

int main(int argc, char** argv) {
    // const std::string param_train_file	= "../data/ml-tag.test.libfm"; // "libfm_test.txt";
    // // const std::string param_train_file	= "../scripts/libfm_test_data_large.txt"; //
    // "libfm_test.txt"; Data train; train.load(param_train_file); for (int i = 0; i < 10; i++) {
    // 	std :: cout << train.target[i] << " ";
    // 	for (int j = 0; j < train.data[i]->size; j++) {
    // 		std::cout << train.data[i]->data[j].id << ":" << train.data[i]->data[j].value << "
    // ";
    // 	}
    // 	std::cout << std :: endl;
    // }
    // fm_model fm(train.num_feature, 8);
    // fm.params.learn_rate = 0.05;
    // fm.params.task = 1;
    // fm.params.min_target = train.min_target;
    // fm.params.max_target = train.max_target;
    auto start_time = std::chrono::steady_clock::now();
    // fm.learn(&train, &train, 2);

    // [[1 0 2]
    // [0 3 0]
    // [4 0 5]]

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    double values[5] = {1, 2, 3, 4, 5};
    int colIdx[5] = {0, 2, 1, 0, 2};
    int rowPtr[4] = {0, 2, 3, 5};


    double* d_A_values;
    int* devrows;
    int* devcols;
    hipMalloc((void**)&d_A_values, 5 * sizeof(double));
    hipMalloc((void**) &devrows, 4 * sizeof(int));
    hipMalloc((void**)&devcols, 5 * sizeof(int));


    hipMemcpy(d_A_values, values, 5 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devrows, rowPtr, 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devcols, colIdx, 5 * sizeof(int), hipMemcpyHostToDevice);


    hipsparseSpMatDescr_t descrA;
    hipsparseCreateCsr(&descrA, 3, 3, 5, devrows, devcols, d_A_values, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);


    // hipMemcpy(a_C, d_A_values, 5 * sizeof(double), hipMemcpyDeviceToHost);wrong
    std:: cout << "Current matrix A" << std::endl;
    //printCudaSparse(descrA);

    double *Ahost = (double*) malloc(sizeof(double) * 5);
    hipMemcpy(Ahost, d_A_values, sizeof(double) * 5, hipMemcpyDeviceToHost);

    for (int i=0; i < 5; i++) {
        std::cout << Ahost[i] << " ";
    }
    std::cout << std::endl;

    // turns out that any modifications to descrA also modifies device ptr d_A_values, so we might not need printCudaSparse



    double values2[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};

    hipsparseDnMatDescr_t descrB;

    double* d_B_values;
    hipMalloc((void**)&d_B_values, 9 * sizeof(double));
    hipMemcpy(d_B_values, values2, 9 * sizeof(double), hipMemcpyHostToDevice);

    hipsparseCreateDnMat(&descrB, 3, 3, 3, d_B_values, HIP_R_64F, HIPSPARSE_ORDER_ROW); //ld is number of rows of matrix

    double* b_C = (double*)malloc(9 * sizeof(double));

    //hipMemcpy(b_C, d_B_values, 9 * sizeof(double), hipMemcpyDeviceToHost); wrong 

    double* d_values;
    int* d_colIdx;
    int* d_rowPtr;

    hipMalloc((void**)&d_values, 5 * sizeof(double));
    hipMalloc((void**)&d_colIdx, 5 * sizeof(int));
    hipMalloc((void**)&d_rowPtr, 4 * sizeof(int));

    hipMemcpy(d_values, values, 5 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, colIdx, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtr, rowPtr, 4 * sizeof(int), hipMemcpyHostToDevice);


    std::cout << "Current matrix B" << std::endl;

    double* Bhost = (double*) malloc(sizeof(double) * 9);
    hipMemcpy(Bhost, d_B_values, sizeof(double) * 9, hipMemcpyDeviceToHost);

    for (int i = 0; i < 9; i++){
        std::cout << Bhost[i] << " " ;
    }
    std::cout << std::endl;

    //printCudaDense(descrB);

    /// double* d_C;
    // hipMalloc((void**)&d_C, 9 * sizeof(double));

    hipsparseDnMatDescr_t descrC;
    double valuesC[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
    double* d_C_values;
    hipMalloc((void**)&d_C_values, 9 * sizeof(double));
    hipMemcpy(d_C_values, valuesC, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipsparseCreateDnMat(&descrC, 3, 3, 3, d_C_values, HIP_R_64F, HIPSPARSE_ORDER_ROW);
    
    std::cout << "Current matrix C" << std::endl;

    double* dHost = (double *) malloc(sizeof(double) * 9);
    hipMemcpy(dHost, d_C_values, sizeof(double) * 9, hipMemcpyDeviceToHost);


       
    for(int i = 0; i < 9; i++) {
        std::cout << dHost[i] << " " ;
    }
    std:: cout << std::endl;

    size_t buffer_size = 0;

    int nnzC = 0;
    int* nnzTotalDevHostPtr = &nnzC;  // what's this for ? 

    const float alpha = 1.0;
    const float beta = 0;
    
    //hipsparseSpMMAlg_t alg = ;

    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, (void*)&alpha, descrA, descrB,
                            (void*)&beta, descrC, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &buffer_size);

    void* buffer = NULL;
    std::cout <<"buffer_size " << buffer_size << std::endl;
    hipMalloc(&buffer, buffer_size);

    hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 (void*)&alpha, descrA, descrB, (void*)&beta, descrC, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, buffer);


    std:: cout << "Result of matmul" <<std::endl;
    //printCudaDense(descrC);    
   
    hipMemcpy(dHost, d_C_values, sizeof(double) * 9, hipMemcpyDeviceToHost);


    std::cout << "DC values " << std:: endl; // this will store all the modified values as well
    for(int i = 0; i < 9; i++) {
        std::cout << dHost[i] << " " ;
    }






    hipFree(d_C_values);

    // hipFree(d_values);
    // hipFree(d_colIdx);
    // hipFree(d_rowPtr);
    // // hipFree(d_C);
    // hipsparseDestroySpMat(descrA);
    // hipsparseDestroyDnMat(descrB);
    // hipsparseDestroyDnMat(descrC);
    // hipsparseDestroy(handle);

    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();
    // Finalize
    std::cout << "Simulation Time = " << seconds << " seconds \n";
    /*
srand ( time(NULL) );
    try {

            const std::string param_task		= "c";
            const std::string param_train_file	= "libfm_test.txt";
            const std::string param_test_file	= "libfm_test.txt";

            double param_init_stdev	= 0.1;
            int param_num_iter	= 100;
            double param_learn_rate	= 0.01;
            const std::string param_method		= "sgd";

            const std::string param_do_sampling	= "do_sampling";
            const std::string param_do_multilevel	= "do_multilevel";
            const std::string param_num_eval_cases  = "num_eval_cases";
            // (1) Load the data
            std::cout << "Loading train...\t" << std::endl;
            Data train(
                    0,
                    ! (!param_method.compare("mcmc")), // no original data for mcmc
                    ! (!param_method.compare("sgd") || !param_method.compare("sgda")) // no
transpose data for sgd, sgda
            );
            train.load(param_train_file);

            std::cout << "Loading test... \t" << std::endl;
            Data test(
                    0,
                    ! (!param_method.compare("mcmc")), // no original data for mcmc
                    ! (!param_method.compare("sgd") || !param_method.compare("sgda")) // no
transpose data for sgd, sgda
            );
            test.load(param_test_file);

            Data* validation = NULL;

            // (2) Setup the factorization machine
            fm_model fm;
            {
        uint num_all_attribute = std::max(train.num_feature, test.num_feature);
                    fm.num_attribute = num_all_attribute;
                    fm.init_stdev = param_init_stdev;
                    // set the number of dimensions in the factorization
                    {
                            std::vector<int> dim(3);
            dim[0] = 1;
            dim[1] = 1;
            dim[2] = 8;
                            assert(dim.size() == 3);
                            fm.k0 = dim[0] != 0;
                            fm.k1 = dim[1] != 0;
                            fm.num_factor = dim[2];
                    }
                    fm.init();

            }

            // (3) Setup the learning method:
            fm_learn* fml;
            if (! param_method.compare("sgd")) {
                    fml = new fm_learn();
                    fml->num_iter = param_num_iter;
            } else {
                    throw "unknown method";
            }
            fml->fm = &fm;
            fml->max_target = train.max_target;
            fml->min_target = train.min_target;
            if (! param_task.compare("r") ) {
                    fml->task = 0;
            } else if (! param_task.compare("c") ) {
                    fml->task = 1;
                    for (uint i = 0; i < train.target.dim; i++) { if (train.target(i) <= 0.0) {
train.target(i) = -1.0; } else {train.target(i) = 1.0; } } for (uint i = 0; i < test.target.dim;
i++) { if (test.target(i) <= 0.0) { test.target(i) = -1.0; } else {test.target(i) = 1.0; } } } else
{ throw "unknown task";
            }

    fml->init();
    // set the regularization; for standard SGD, groups are not supported
    {
        std::vector<double> reg(3);
        reg[2] == 0.01;
        assert((reg.size() == 0) || (reg.size() == 1) || (reg.size() == 3));
        if (reg.size() == 0) {
            fm.reg0 = 0.0;
            fm.regw = 0.0;
            fm.regv = 0.0;
        } else if (reg.size() == 1) {
            fm.reg0 = reg[0];
            fm.regw = reg[0];
            fm.regv = reg[0];
        } else {
            fm.reg0 = reg[0];
            fm.regw = reg[1];
            fm.regv = reg[2];
        }
    }
            {
                    std::vector<double> lr(1, param_learn_rate);
                    assert((lr.size() == 1) || (lr.size() == 3));
                    if (lr.size() == 1) {
                            fml->learn_rate = lr[0];
                            fml->learn_rates.init(lr[0]);
                    } else {
                            fml->learn_rate = 0;
                            fml->learn_rates(0) = lr[0];
                            fml->learn_rates(1) = lr[1];
                            fml->learn_rates(2) = lr[2];
                    }
            }

            auto start_time = std::chrono::steady_clock::now();

            // () learn
            fml->learn(&train, &test);

            // () Prediction at the end  (not for mcmc and als)
            std::cout << "Final\t" << "Train=" << fml->evaluate(&train) << "\tTest=" <<
fml->evaluate(&test) << std::endl;

            auto end_time = std::chrono::steady_clock::now();
            std::chrono::duration<double> diff = end_time - start_time;
            double seconds = diff.count();

            // Finalize
            std::cout << "Simulation Time = " << seconds << " seconds \n";


    } catch (std::string &e) {
            std::cerr << std::endl << "ERROR: " << e << std::endl;
    } catch (char const* &e) {
            std::cerr << std::endl << "ERROR: " << e << std::endl;
    }
    */
}
