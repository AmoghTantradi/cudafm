#include "hip/hip_runtime.h"
#include "fm.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iomanip>
#include "util/random.h"

#define NUM_THREADS 256


fm_model::fm_model(int n, int k) {

	init_mean = 0;
	init_stdev = 0.01;

	double* vTemp = (double*)malloc(n*k * sizeof(double));//initialize random, cuda malloc and copy into v
	for (int i = 0; i < n*k; i++) {
		vTemp[i] = ran_gaussian(init_mean, init_stdev);
	}
	hipMalloc((void**)&v, sizeof(double) * n * k);
	hipMemcpy(v, vTemp, sizeof(double) * n * k, hipMemcpyHostToDevice);
	free(vTemp);

	double* wTemp = (double*)malloc(n * sizeof(double));//initialize random, cuda malloc and copy into v
	for (int i = 0; i < n; i++) {
		wTemp[i] = ran_gaussian(init_mean, init_stdev);
	}
	hipMalloc((void**)&w, sizeof(double) * n);
	hipMemcpy(w, wTemp, sizeof(double) * n, hipMemcpyHostToDevice);
	free(wTemp);
	
	double w0Temp = 0;
	hipMalloc((void**)&w0, sizeof(double));
	hipMemcpy(w0, &w0Temp, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&m_sum, n * sizeof(double));
	hipMalloc((void**)&m_sum_sqr, n * sizeof(double));
	//no cuda
	params.num_attribute = n;
	params.num_factor = k;
}


// X data must be stored as vector of  where the sparse entry resides in cuda memory, Y data is stored as vector of doubles


//done
double fm_model::evaluate(Data* data) {
  assert(data.data != NULL);
  if (params.task == 0) {
    return evaluate_regression(data);
  } else if (params.task == 1) {
    return evaluate_classification(data);
  } else {
    throw "unknown task";
  }
}

//done
void fm_model::learn(Data* train, Data* test, int num_iter) {
    std::cout << "learnrate=" << params.learn_rate << std::endl;
	std::cout << "#iterations=" << num_iter << std::endl;
	
    std::cout.flush();
    std::cout << "SGD: DON'T FORGET TO SHUFFLE THE ROWS IN TRAINING DATA TO GET THE BEST RESULTS." << std::endl;
	// SGD
	for (int i = 0; i < train->data.size(); i++) {
		sparse_row_v<DATA_FLOAT>* sample;
		int memsize = sizeof(sparse_row_v<DATA_FLOAT>) + train->data[i]->size*sizeof(sparse_entry<DATA_FLOAT>);
		hipMalloc((void**)&sample, memsize);
		hipMemcpy(sample, train->data[i], memsize, hipMemcpyHostToDevice);
		//free(train->data[i]);
		train->data[i] = sample;
	}

	hipMalloc((void**)&cuda_args, sizeof(cudaArgs));
	cudaArgs args;
	args.w0 = w0;
	args.w = w;
	args.v = v;
	args.params = params;
	hipMalloc((void**)&ret, sizeof(double));
	args.ret = ret;
	hipMemcpy(cuda_args, &args, sizeof(cudaArgs), hipMemcpyHostToDevice);
	

    for (int i = 0; i < num_iter; i++) {
		// new version


		// old version
        for (int j = 0; j < train->data.size(); j++) {
        	double p = predict(train->data[j], m_sum, m_sum_sqr);
			//std:: cout << p << "\n";
			// double p = 0;
        	double mult = 0;
			if (params.task == 0) {
				p = std::min(params.max_target, p);
				p = std::max(params.min_target, p);
				mult = -(train->target[j]-p);
			} else if (params.task == 1) {
				mult = -train->target[j]*(1.0-1.0/(1.0+exp(-train->target[j]*p)));
			}
        	SGD(train->data[j], mult, m_sum);

			// predict(train -> data[j], m_sum, m_sum_sqr, args);
        }
        double rmse_train = evaluate(train);
		std::cout << rmse_train << "\n";

		//std::cout << i << "\n";
        //double rmse_test = evaluate(test);
        //std::cout << "#Iter=" << std::setw(3) << i << "\tTrain=" << rmse_train << "\tTest=" << rmse_test << std::endl;
    }

}


__global__ void cudaSGD(sparse_row_v<DATA_FLOAT>* x, const double multiplier, double *sum, cudaArgs* args) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid == 0) {
		*args->w0 -= args->params.learn_rate * (multiplier);
	}
	if (tid >= x->size * args -> params.num_factor)
        return;

	/*
	for (uint i = 0; i < x->size; i++) {
		args->w[i] -= args->params.learn_rate * (multiplier * x->data[i].value);
	}
	*/
	int nf = args -> params.num_factor;

	int f = int(tid / nf); // corresponds to f
	int idx = tid - f * nf; // corresponds to i


	if(f == 0) {
		args->w[idx] -= args->params.learn_rate * (multiplier * x->data[idx].value);
	}

	double& v1 = args->v[f * args->params.num_attribute + x->data[idx].id];
	double grad = sum[f] * x->data[idx].value - v1 * x->data[idx].value * x->data[idx].value;
	v1 -= args->params.learn_rate * (multiplier * grad);


	/**
	for (int f = 0; f < args->params.num_factor; f++) {
		for (uint i = 0; i < x->size; i++) {
			double& v1 = args->v[f*args->params.num_attribute + x->data[i].id];
			double grad = sum[f] * x->data[i].value - v1 * x->data[i].value * x->data[i].value; 
			v1 -= args->params.learn_rate * (multiplier * grad);
		}
	}
	*/
}
//done
void fm_model::SGD(sparse_row_v<DATA_FLOAT>* x, const double multiplier, double *sum) {
	/*
	*w0 -= params.learn_rate * (multiplier);
	for (uint i = 0; i < x->size; i++) {
		w[i] -= params.learn_rate * (multiplier * x->data[i].value);
	}
	for (int f = 0; f < params.num_factor; f++) {
		for (uint i = 0; i < x->size; i++) {
			double& v1 = v[f*params.num_attribute + x->data[i].id];
			double grad = sum[f] * x->data[i].value - v1 * x->data[i].value * x->data[i].value; 
			v1 -= params.learn_rate * (multiplier * grad);
		}
	}
	*/
	cudaSGD<<<1, NUM_THREADS>>>(x, multiplier, sum, cuda_args);
}


/*
__device__ void cudaDotProd(double *v, sparse_entry<DATA_FLOAT> x, ) {
	atomicAdd(, v[x.id] * x.value);
}
*/

__global__ void cudaPredict(sparse_row_v<DATA_FLOAT>* x, double* sum, double* sum_sqr, cudaArgs* args) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int nf = args -> params.num_factor;
	if (tid >= x->size * nf) {
		return;
	}

	//tid = face_value * num_factor + i where i < x.size 
	int f = int(tid / nf); // corresponds to f
	int idx = tid - f * nf; // corresponds to i

	if (f == 0) {
		atomicAdd(args->ret, args -> w[x->data[idx].id] * x->data[idx].value);
	} // adds the linear terms 
	
	double val = args -> v[f *args->params.num_attribute + x->data[idx].id] * x->data[idx].value;

	atomicAdd(&sum[f], val); // stores in sum[Faceval]
	atomicAdd(&sum_sqr[f], val * val);
}

__global__ void aggregate(double * ret, double * sum, double * sum_sqr, cudaArgs * args) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= args -> params.num_factor) {
		return;
	}
	//ret should be initialized to 0 first 
	atomicAdd(ret, 0.5 * (sum[tid] * sum[tid] - sum_sqr[tid]));
}

double fm_model::predict(sparse_row_v<DATA_FLOAT>* x) {
	return predict(x, m_sum, m_sum_sqr);
}

double fm_model::predict(sparse_row_v<DATA_FLOAT>* x, double* sum, double* sum_sqr) {
	double pred;
	hipMemcpy(ret, w0, sizeof(double), hipMemcpyDeviceToDevice);

	hipMemset(sum, 0, sizeof(int) * params.num_factor);
	hipMemset(sum_sqr, 0, sizeof(int) * params.num_factor);

	cudaPredict<<<1, NUM_THREADS>>>(x, sum, sum_sqr, cuda_args);
	aggregate<<<1, NUM_THREADS>>>(ret, sum, sum_sqr, cuda_args);

	hipMemcpy(&pred, ret, sizeof(double), hipMemcpyDeviceToHost);

	return pred;
}

//predict
void fm_model::predict(Data* data, double* out) {
  for (int i = 0; i < data->data.size(); i++) {
    double p = predict(data->data[i]);
    if (params.task == 0 ) {
      p = std::min(params.max_target, p);
      p = std::max(params.min_target, p);
    } else if (params.task == 1) {
      p = 1.0/(1.0 + exp(-p));
    } else {
      throw "task not supported";
    }
    out[i] = p;
  }
}

double fm_model::evaluate_classification(Data* data) {
  int num_correct = 0;
  for (int i = 0; i < data->data.size(); i++) {
    double p = predict(data->data[i]);
    if (((p >= 0) && (data->target[i] >= 0)) || ((p < 0) && (data->target[i] < 0))) {
      num_correct++;
    }
  }

  return (double) num_correct / (double) data->data.size();
}

double fm_model::evaluate_regression(Data* data) {
  double rmse_sum_sqr = 0;
  double mae_sum_abs = 0;
  for (int i = 0; i < data->data.size(); i++) {
    double p = predict(data->data[i]);
    p = std::min(params.max_target, p);
    p = std::max(params.min_target, p);
    double err = p - data->target[i];
    rmse_sum_sqr += err*err;
    mae_sum_abs += std::abs((double)err);
  }

  return std::sqrt(rmse_sum_sqr/data->data.size());
}