#include "hip/hip_runtime.h"
#include "fm.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iomanip>
#include "util/random.h"

#define NUM_THREADS 256


fm_model::fm_model(int n, int k) {
	/*
	hipMalloc(&num_attribute, sizeof(int));
	hipMalloc(&num_factor, sizeof(int));

	hipMalloc(&w, sizeof(double) * n);

	

	hipMalloc(&w0, sizeof(double));
	
	*num_attribute = n; //do in cuda memory
	*num_factor = k; //do in cuda memory
	init_mean = 0;
	init_stdev = 0.01;

	double* vTemp = (double*)malloc(n*k * sizeof(double));//initialize random, cuda malloc and copy into v
	for (int i = 0; i < n*k; i++) {
		vTemp[i] = ran_gaussian(init_mean, init_stdev);
	}

	
	double* wTemp = (double*)malloc(n * sizeof(double));//initialize random, cuda malloc and copy into v
	for (int i = 0; i < n; i++) {
		wTemp[i] = ran_gaussian(init_mean, init_stdev);
	}
	
	hipMemcpy(w, wTemp, sizeof(double) * n, hipMemcpyHostToDevice);
	
	free(wTemp);
	double w0Temp = 0; //copy to cuda
	hipMemcpy(w0, &w0Temp, sizeof(double), hipMemcpyHostToDevice);
	//cuda malloc m_sum and m_sum_sqr
	learn_rate = 0.01;
	*/
	init_mean = 0;
	init_stdev = 0.01;

	double* vTemp = (double*)malloc(n*k * sizeof(double));//initialize random, cuda malloc and copy into v
	for (int i = 0; i < n*k; i++) {
		vTemp[i] = ran_gaussian(init_mean, init_stdev);
	}
	hipMalloc((void**)&v, sizeof(double) * n * k);
	hipMemcpy(v, vTemp, sizeof(double) * n * k, hipMemcpyHostToDevice);
	free(vTemp);

	double* wTemp = (double*)malloc(n * sizeof(double));//initialize random, cuda malloc and copy into v
	for (int i = 0; i < n; i++) {
		wTemp[i] = ran_gaussian(init_mean, init_stdev);
	}
	hipMalloc((void**)&w, sizeof(double) * n);
	hipMemcpy(w, wTemp, sizeof(double) * n, hipMemcpyHostToDevice);
	free(wTemp);
	
	double w0Temp = 0;
	hipMalloc((void**)&w0, sizeof(double));
	hipMemcpy(w0, &w0Temp, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&m_sum, n * sizeof(double));
	hipMalloc((void**)&m_sum_sqr, n * sizeof(double));
	//no cuda
	params.num_attribute = n;
	params.num_factor = k;
}

/*
__global__ void cudaPredict(sparse_entry<DATA_FLOAT>* x, int xsize, int n, double* w0, double* w, double* v, double * pred) {

}
*/

__global__ void cudaPredict(sparse_row_v<DATA_FLOAT>* x, double* sum, double* sum_sqr, cudaArgs* args) {
	double pred = 0;
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid == 0) {
		pred += *args->w0;
	}
	if (tid >= x->size)
        return;
	for (uint i = 0; i < x->size; i++) {
		pred += args->w[x->data[i].id] * x->data[i].value;
	}
	for (int f = 0; f < args->params.num_factor; f++) {
		sum[f] = 0;
		sum_sqr[f] = 0;
		for (uint i = 0; i < x->size; i++) {
			double d = args->v[f*args->params.num_attribute + x->data[i].id] * x->data[i].value;
			sum[f] += d;
			sum_sqr[f] += d*d;
		}
		pred += 0.5 * (sum[f]*sum[f] - sum_sqr[f]);
	}
	*args->ret = pred;
}

/*
double fm_model::predict(sparse_entry<DATA_FLOAT>* x, int xsize) {
	return predict(x, xsize, m_sum, m_sum_sqr);
}

double fm_model::predict(sparse_entry<DATA_FLOAT>* x, int xsize, double* sum, double* sum_sqr) {
	
	double* pred; //cudamalloc this
	double hostPred; 
	hipMalloc(&pred, sizeof(double));
	
	int blks = (xsize + NUM_THREADS-1)/NUM_THREADS;
	cudaPredict<<<blks, NUM_THREADS>>>(x, xsize, num_attribute, num_factor, w0, w, v, sum, sum_sqr, pred);
	//bring pred to host
	hipMemcpy(&hostPred, pred, sizeof(long), hipMemcpyDeviceToHost);
	hipFree(pred);
	
	return hostPred;
}

*/

// X data must be stored as vector of  where the sparse entry resides in cuda memory, Y data is stored as vector of doubles


//done
double fm_model::evaluate(Data* data) {
  assert(data.data != NULL);
  if (params.task == 0) {
    return evaluate_regression(data);
  } else if (params.task == 1) {
    return evaluate_classification(data);
  } else {
    throw "unknown task";
  }
}

/*
__global__ void printSparseRow(sparse_row_v<DATA_FLOAT>*vi) {
	printf("hi %d\n", vi->size);
	for (int j = 0; j < vi->size; j++) {
		printf("%d:%f ", vi->data[j].id, vi->data[j].value); 
	}
	printf("\n");
}
*/

//done
void fm_model::learn(Data* train, Data* test, int num_iter) {
    std::cout << "learnrate=" << params.learn_rate << std::endl;
	std::cout << "#iterations=" << num_iter << std::endl;
	
    std::cout.flush();
    std::cout << "SGD: DON'T FORGET TO SHUFFLE THE ROWS IN TRAINING DATA TO GET THE BEST RESULTS." << std::endl;
	// SGD
	for (int i = 0; i < train->data.size(); i++) {
		sparse_row_v<DATA_FLOAT>* sample;
		int memsize = sizeof(sparse_row_v<DATA_FLOAT>) + train->data[i]->size*sizeof(sparse_entry<DATA_FLOAT>);
		hipMalloc((void**)&sample, memsize);
		hipMemcpy(sample, train->data[i], memsize, hipMemcpyHostToDevice);
		//free(train->data[i]);
		train->data[i] = sample;
	}

	hipMalloc((void**)&cuda_args, sizeof(cudaArgs));
	cudaArgs args;
	args.w0 = w0;
	args.w = w;
	args.v = v;
	args.params = params;
	hipMalloc((void**)&ret, sizeof(double));
	args.ret = ret;
	hipMemcpy(cuda_args, &args, sizeof(cudaArgs), hipMemcpyHostToDevice);
	
    for (int i = 0; i < num_iter; i++) {

        for (int j = 0; j < train->data.size(); j++) {
        	double p = predict(train->data[j], m_sum, m_sum_sqr);
			//std:: cout << p << "\n";
			//double p = 0;
        	double mult = 0;
			if (params.task == 0) {
				p = std::min(params.max_target, p);
				p = std::max(params.min_target, p);
				mult = -(train->target[j]-p);
			} else if (params.task == 1) {
				mult = -train->target[j]*(1.0-1.0/(1.0+exp(-train->target[j]*p)));
			}
        	SGD(train->data[j], mult, m_sum);

        }
        //double rmse_train = evaluate(train);
		//std::cout << rmse_train << "\n";
		//std::cout << i << "\n";
        //double rmse_test = evaluate(test);
        //std::cout << "#Iter=" << std::setw(3) << i << "\tTrain=" << rmse_train << "\tTest=" << rmse_test << std::endl;
    }
}

__global__ void cudaSGD(sparse_row_v<DATA_FLOAT>* x, const double multiplier, double *sum, cudaArgs* args) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid == 0) {
		*args->w0 -= args->params.learn_rate * (multiplier);
	}
	if (tid >= x->size)
        return;
	for (uint i = 0; i < x->size; i++) {
		args->w[i] -= args->params.learn_rate * (multiplier * x->data[i].value);
	}
	for (int f = 0; f < args->params.num_factor; f++) {
		for (uint i = 0; i < x->size; i++) {
			double& v1 = args->v[f*args->params.num_attribute + x->data[i].id];
			double grad = sum[f] * x->data[i].value - v1 * x->data[i].value * x->data[i].value; 
			v1 -= args->params.learn_rate * (multiplier * grad);
		}
	}
}
//done
void fm_model::SGD(sparse_row_v<DATA_FLOAT>* x, const double multiplier, double *sum) {
	/*
	*w0 -= params.learn_rate * (multiplier);
	for (uint i = 0; i < x->size; i++) {
		w[i] -= params.learn_rate * (multiplier * x->data[i].value);
	}
	for (int f = 0; f < params.num_factor; f++) {
		for (uint i = 0; i < x->size; i++) {
			double& v1 = v[f*params.num_attribute + x->data[i].id];
			double grad = sum[f] * x->data[i].value - v1 * x->data[i].value * x->data[i].value; 
			v1 -= params.learn_rate * (multiplier * grad);
		}
	}
	*/
	cudaSGD<<<1, NUM_THREADS>>>(x, multiplier, sum, cuda_args);
}

double fm_model::predict(sparse_row_v<DATA_FLOAT>* x) {
	return predict(x, m_sum, m_sum_sqr);
}

double fm_model::predict(sparse_row_v<DATA_FLOAT>* x, double* sum, double* sum_sqr) {
	/*
	double result = 0;
	result += *w0;
	for (uint i = 0; i < x->size; i++) {
		assert(x.data[i].id < num_attribute);
		result += w[x->data[i].id] * x->data[i].value;
	}
	for (int f = 0; f < params.num_factor; f++) {
		sum[f] = 0;
		sum_sqr[f] = 0;
		for (uint i = 0; i < x->size; i++) {
			double d = v[f*params.num_attribute + x->data[i].id] * x->data[i].value;
			sum[f] += d;
			sum_sqr[f] += d*d;
		}
		result += 0.5 * (sum[f]*sum[f] - sum_sqr[f]);
	}
	return result;
	*/
	double pred;
	cudaPredict<<<1, NUM_THREADS>>>(x, sum, sum_sqr, cuda_args);
	hipMemcpy(&pred, ret, sizeof(double), hipMemcpyDeviceToHost);
	return pred;
}

//predict
void fm_model::predict(Data* data, double* out) {
  for (int i = 0; i < data->data.size(); i++) {
    double p = predict(data->data[i]);
    if (params.task == 0 ) {
      p = std::min(params.max_target, p);
      p = std::max(params.min_target, p);
    } else if (params.task == 1) {
      p = 1.0/(1.0 + exp(-p));
    } else {
      throw "task not supported";
    }
    out[i] = p;
  }
}

double fm_model::evaluate_classification(Data* data) {
  int num_correct = 0;
  for (int i = 0; i < data->data.size(); i++) {
    double p = predict(data->data[i]);
    if (((p >= 0) && (data->target[i] >= 0)) || ((p < 0) && (data->target[i] < 0))) {
      num_correct++;
    }
  }

  return (double) num_correct / (double) data->data.size();
}

double fm_model::evaluate_regression(Data* data) {
  double rmse_sum_sqr = 0;
  double mae_sum_abs = 0;
  for (int i = 0; i < data->data.size(); i++) {
    double p = predict(data->data[i]);
    p = std::min(params.max_target, p);
    p = std::max(params.min_target, p);
    double err = p - data->target[i];
    rmse_sum_sqr += err*err;
    mae_sum_abs += std::abs((double)err);
  }

  return std::sqrt(rmse_sum_sqr/data->data.size());
}